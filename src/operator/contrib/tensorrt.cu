
#include <hip/hip_runtime.h>
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2018 by Contributors
 * \file trt.cu
 * \brief TensorRT GPU operation
 * \author Marek Kolodziej, Clement Fuji Tsang
*/

#if MXNET_USE_TENSORRT

#include "./tensorrt-inl.h"

namespace mxnet {
namespace op {

#define CHECK_CUDART(x) do { \
  cudaError_t res = (x); \
  if (res != cudaSuccess) { \
    fprintf(stderr, "CUDART: %s = %d (%s) at (%s:%d)\n", \
      #x, res, cudaGetErrorString(res), __FILE__, __LINE__); \
    exit(1); \
  } \
} while (0)

void TRTCompute(const OpStatePtr& state, const OpContext& ctx,
                     const std::vector<TBlob>& inputs, const std::vector<OpReqType>& req,
                     const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  using namespace mshadow::expr;

  Stream<gpu>* s = ctx.get_stream<gpu>();
  cudaStream_t cuda_s = Stream<gpu>::GetStream(s);
  const auto& param = state.get_state<TRTEngineParam>();
  std::vector<void*> bindings;
  bindings.reserve(param.binding_map.size());
  for (auto& p : param.binding_map) {
    if (p.second == tensorrt::TypeIO::Inputs) {
      bindings.emplace_back(inputs[p.first].dptr_);
    } else {
      bindings.emplace_back(outputs[p.first].dptr_);
    }
  }

  const int batch_size = static_cast<int>(inputs[0].shape_[0]);
  param.trt_executor->enqueue(batch_size, bindings.data(), cuda_s, nullptr);
  CHECK_CUDART(cudaStreamSynchronize(cuda_s));
}

NNVM_REGISTER_OP(_trt_op)
.set_attr<FStatefulCompute>("FStatefulCompute<gpu>", TRTCompute);

}  // namespace op
}  // namespace mxnet

#endif  // MXNET_USE_TENSORRT
