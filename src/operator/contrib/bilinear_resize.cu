#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */
/*!
 * Copyright (c) 2018 by Contributors
 * \file bilinear_resize.cu
 * \brief bilinear resize operator
 * \author Hang Zhang
*/
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include "bilinear_resize-inl.h"

namespace mxnet {
namespace op {

using namespace mshadow;

template<typename In, typename Out>
struct ScalarConvert {
  static __host__ __device__ __forceinline__ Out to(const In v) { return (Out) v; }
};


// The maximum number of threads in a block
static const unsigned MAX_BLOCK_SIZE = 512U;

// Number of threads in a block given an input size up to MAX_BLOCK_SIZE
static unsigned getNumThreads(int nElem, const bool smaller) {
  unsigned threadSizes[5] = {32, 64, 128, 256, MAX_BLOCK_SIZE};
  const int maxi = smaller ? 4 : 5;
  for (int i = 0; i != maxi; ++i) {
    if (static_cast<unsigned>(nElem) <= threadSizes[i]) {
      return threadSizes[i];
    }
  }
  return smaller ? (MAX_BLOCK_SIZE >> 1) : MAX_BLOCK_SIZE;
}

template<typename xpu, typename Dtype, typename Acctype>
__global__ void caffe_gpu_interp2_kernel(const int n,
    const Acctype rheight, const Acctype rwidth,
    const Tensor<xpu, 4, Dtype> data1,
    Tensor<xpu, 4, Dtype> data2) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.size(0);
  const int channels = data1.size(1);
  const int height1 = data1.size(2);
  const int width1 = data1.size(3);
  const int height2 = data2.size(2);
  const int width2 = data2.size(3);

  if (index < n) {
    const int w2 = index % width2;  // 0:width2-1
    const int h2 = index / width2;  // 0:height2-1
    // special case: just copy
    if (height1 == height2 && width1 == width2) {
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++) {
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data1[n][c][h1][w1];
          data2[n][c][h2][w2] = val;
        }
      }
      return;
    }
    //
    const Acctype h1r = rheight * h2;
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const Acctype h1lambda = h1r - h1;
    const Acctype h0lambda = Acctype(1) - h1lambda;
    //
    const Acctype w1r = rwidth * w2;
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++) {
        for (int c = 0; c < channels; ++c) {
        const Acctype val = h0lambda * (w0lambda * data1[n][c][h1][w1]
                            + w1lambda * data1[n][c][h1][w1+w1p])
                            + h1lambda * (w0lambda * data1[n][c][h1+h1p][w1]
                            + w1lambda * data1[n][c][h1+h1p][w1+w1p]);
        data2[n][c][h2][w2] = ScalarConvert<Acctype, Dtype>::to(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template<typename xpu, typename Dtype, typename Acctype>
__global__ void caffe_gpu_interp2_kernel_backward(const int n,
    const Acctype rheight, const Acctype rwidth,
    Tensor<xpu, 4, Dtype> data1, const Tensor<xpu, 4, Dtype> data2) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.size(0);
  const int channels = data1.size(1);
  const int height1 = data1.size(2);
  const int width1 = data1.size(3);
  const int height2 = data2.size(2);
  const int width2 = data2.size(3);
  if (index < n) {
    const int w2 = index % width2;  // 0:width2-1
    const int h2 = index / width2;  // 0:height2-1
    // special case: just copy
    if (height1 == height2 && width1 == width2) {
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++) {
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data2[n][c][h1][w1];
          data1[n][c][h2][w2] += val;
        }
      }
      return;
    }
    //
    const Acctype h1r = rheight * h2;
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const Acctype h1lambda = h1r - h1;
    const Acctype h0lambda = Acctype(1) - h1lambda;
    //
    const Acctype w1r = rwidth * w2;
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++) {
      for (int c = 0; c < channels; ++c) {
        const Dtype d2val = data2[n][c][h2][w2];
        atomicAdd(&data1[n][c][h1][w1],
                  ScalarConvert<Acctype, Dtype>::to(h0lambda * w0lambda * d2val));
        atomicAdd(&data1[n][c][h1][w1+w1p],
                  ScalarConvert<Acctype, Dtype>::to(h0lambda * w1lambda * d2val));
        atomicAdd(&data1[n][c][h1+h1p][w1],
                  ScalarConvert<Acctype, Dtype>::to(h1lambda * w0lambda * d2val));
        atomicAdd(&data1[n][c][h1+h1p][w1+w1p],
                  ScalarConvert<Acctype, Dtype>::to(h1lambda * w1lambda * d2val));
      }
    }
  }
}

template<typename xpu, typename DType, typename AccReal>
void SpatialUpSamplingBilinearUpdateOutput(mshadow::Stream<gpu> *s,
                                           const std::vector<TBlob> &input,
                                           const std::vector<TBlob> &output) {
  Tensor<xpu, 4, DType> idata = input[0].get<xpu, 4, DType>(s);
  Tensor<xpu, 4, DType> odata = output[0].get<xpu, 4, DType>(s);
  int outputHeight = odata.size(2);
  int outputWidth = odata.size(3);
  int inputHeight = idata.size(2);
  int inputWidth = idata.size(3);

  const AccReal rheight = (outputHeight > 1) ? (AccReal)(inputHeight - 1)/
                         (outputHeight - 1) : AccReal(0);
  const AccReal rwidth = (outputWidth > 1) ? (AccReal)(inputWidth - 1)/
                         (outputWidth - 1) : AccReal(0);
  const int num_kernels = outputHeight * outputWidth;
  const int num_threads = getNumThreads(inputHeight*inputWidth, false);
  dim3 blocks(static_cast<int>(num_kernels / num_threads) + 1);
  dim3 threads(num_threads);
  hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
  caffe_gpu_interp2_kernel<xpu, DType, AccReal>
  <<<blocks, threads , 0, stream>>>(
    num_kernels, rheight, rwidth, idata, odata);
  MSHADOW_CUDA_POST_KERNEL_CHECK(SpatialUpSamplingBilinearUpdateOutput);
}

template<typename xpu, typename DType, typename AccReal>
void SpatialUpSamplingBilinearUpdateGradInput(mshadow::Stream<gpu> *s,
                                              const std::vector<TBlob> &input,
                                              const std::vector<TBlob> &output) {
  Tensor<xpu, 4, DType> data1 = output[0].get<xpu, 4, DType>(s);
  Tensor<xpu, 4, DType> data2 = input[0].get<xpu, 4, DType>(s);
  int height1 = data1.size(2);
  int width1 = data1.size(3);
  int height2 = data2.size(2);
  int width2 = data2.size(3);
  const AccReal rheight = (height2 > 1) ? (AccReal)(height1 - 1)/(height2 - 1) : AccReal(0);
  const AccReal rwidth = (width2 > 1) ? (AccReal)(width1 - 1) / (width2 - 1) : AccReal(0);
  const int num_kernels = height2 * width2;
  const int num_threads = getNumThreads(height1*width1, false);
  dim3 blocks(static_cast<int>(num_kernels / num_threads) + 1);
  dim3 threads(num_threads);
  hipStream_t stream = mshadow::Stream<gpu>::GetStream(s);
  caffe_gpu_interp2_kernel_backward<xpu, DType, AccReal>
  <<<blocks, threads, 0, stream>>>(
    num_kernels, rheight, rwidth, data1, data2);
  MSHADOW_CUDA_POST_KERNEL_CHECK(SpatialUpSamplingBilinearUpdateGradInput);
}

NNVM_REGISTER_OP(_contrib_BilinearResize2D)
.set_attr<FCompute>("FCompute<gpu>", BilinearSampleOpForward<gpu>);

NNVM_REGISTER_OP(_backward_contrib_BilinearResize2D)
.set_attr<FCompute>("FCompute<gpu>", BilinearSampleOpBackward<gpu>);

}  // namespace op
}  // namespace mxnet
