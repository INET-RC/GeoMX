/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

 /*!
  *  Copyright (c) 2017 by Contributors
  * \file bounding_box.cu
  * \brief Bounding box util functions and operators
  * \author Joshua Zhang
  */

#include "./bounding_box-inl.cuh"
#include "./bounding_box-inl.h"
#include "../elemwise_op_common.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(_contrib_box_nms)
.set_attr<FCompute>("FCompute<gpu>", BoxNMSForward<gpu>);

NNVM_REGISTER_OP(_backward_contrib_box_nms)
.set_attr<FCompute>("FCompute<gpu>", BoxNMSBackward<gpu>);

NNVM_REGISTER_OP(_contrib_box_iou)
.set_attr<FCompute>("FCompute<gpu>", BoxOverlapForward<gpu>);

NNVM_REGISTER_OP(_backward_contrib_box_iou)
.set_attr<FCompute>("FCompute<gpu>", BoxOverlapBackward<gpu>);

NNVM_REGISTER_OP(_contrib_bipartite_matching)
.set_attr<FCompute>("FCompute<gpu>", BipartiteMatchingForward<gpu>);

NNVM_REGISTER_OP(_backward_contrib_bipartite_matching)
.set_attr<FCompute>("FCompute<gpu>", BipartiteMatchingBackward<gpu>);
}  // namespace op
}  // namespace mxnet
