#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file elemwise_unary_op_trig.cu
 * \brief GPU Implementation of unary trigonometric function.
 */
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {

// sin
NNVM_REGISTER_OP(sin)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::sin>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::sin>);

NNVM_REGISTER_OP(_backward_sin)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::sin_grad> >);

// cos
NNVM_REGISTER_OP(cos)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::cos>);

NNVM_REGISTER_OP(_backward_cos)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::cos_grad> >);

// tan
NNVM_REGISTER_OP(tan)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::tan>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::tan>);

NNVM_REGISTER_OP(_backward_tan)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::tan_grad> >);

// arcsin
NNVM_REGISTER_OP(arcsin)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arcsin>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::arcsin>);

NNVM_REGISTER_OP(_backward_arcsin)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arcsin_grad> >);

// arccos
NNVM_REGISTER_OP(arccos)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arccos>);

NNVM_REGISTER_OP(_backward_arccos)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arccos_grad> >);

// arctan
NNVM_REGISTER_OP(arctan)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arctan>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::arctan>);

NNVM_REGISTER_OP(_backward_arctan)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arctan_grad> >);

// degrees
NNVM_REGISTER_OP(degrees)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::degrees>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::degrees>);

NNVM_REGISTER_OP(_backward_degrees)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::degrees_grad> >);

// radians
NNVM_REGISTER_OP(radians)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::radians>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::radians>);

NNVM_REGISTER_OP(_backward_radians)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::radians_grad> >);

// cosh
NNVM_REGISTER_OP(cosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::cosh>);

NNVM_REGISTER_OP(_backward_cosh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::cosh_grad> >);

// sinh
NNVM_REGISTER_OP(sinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::sinh>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::sinh>);

NNVM_REGISTER_OP(_backward_sinh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::sinh_grad> >);

// tanh
NNVM_REGISTER_OP(tanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::tanh>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::tanh>);

NNVM_REGISTER_OP(_backward_tanh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::tanh_grad> >);

// arcsinh
NNVM_REGISTER_OP(arcsinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arcsinh>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::arcsinh>);

NNVM_REGISTER_OP(_backward_arcsinh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arcsinh_grad> >);

// arccosh
NNVM_REGISTER_OP(arccosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arccosh>);

NNVM_REGISTER_OP(_backward_arccosh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arccosh_grad> >);

// arctanh
NNVM_REGISTER_OP(arctanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arctanh>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::arctanh>);

NNVM_REGISTER_OP(_backward_arctanh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arctanh_grad> >);

}  // namespace op
}  // namespace mxnet
