#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file elemwise_unary_op_basic.cu
 * \brief GPU Implementation of unary functions.
 */
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(relu)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::relu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::relu>);

NNVM_REGISTER_OP(_backward_relu)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::relu_grad>>);

NNVM_REGISTER_OP(sigmoid)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::sigmoid>);

NNVM_REGISTER_OP(_backward_sigmoid)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::sigmoid_grad>>);

NNVM_REGISTER_OP(hard_sigmoid)
.set_attr<FCompute>("FCompute<gpu>", HardSigmoidForward<gpu>);

NNVM_REGISTER_OP(_backward_hard_sigmoid)
.set_attr<FCompute>("FCompute<gpu>", HardSigmoidBackward<gpu>);

// softsign
NNVM_REGISTER_OP(softsign)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::softsign>);

NNVM_REGISTER_OP(_backward_softsign)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::softsign_grad>>);

// erf
NNVM_REGISTER_OP(erf)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::erf>);

NNVM_REGISTER_OP(_backward_erf)
.set_attr<FCompute>("FCompute<gpu>",
                    ElemwiseBinaryOp::Compute<gpu, unary_bwd<mshadow_op::erf_grad>>);

// copy
NNVM_REGISTER_OP(_copy)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::IdentityComputeEx<gpu>);

NNVM_REGISTER_OP(_backward_copy)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(BlockGrad)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(make_loss)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::IdentityComputeEx<gpu>);

// identity output as first input, but attributes are constrainted to be like rhs
NNVM_REGISTER_OP(_identity_with_attr_like_rhs)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::IdentityComputeFirstItemEx<gpu>);

NNVM_REGISTER_OP(reshape_like)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

void ShapeComputeGPU(const nnvm::NodeAttrs& attrs,
                     const OpContext& ctx,
                     const std::vector<TBlob>& inputs,
                     const std::vector<OpReqType>& req,
                     const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  CHECK_EQ(inputs.size(), 1U);
  CHECK_EQ(outputs.size(), 1U);
  CHECK_EQ(req.size(), 1U);
  const TBlob& in_data = inputs[0];
  const TBlob& out_data = outputs[0];
  mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
  hipMemcpyAsync(out_data.dptr_,
                  in_data.shape_.data(),
                  in_data.ndim() * sizeof(int64_t),
                  hipMemcpyHostToDevice,
                  mshadow::Stream<gpu>::GetStream(s));
}

NNVM_REGISTER_OP(shape_array)
.set_attr<FCompute>("FCompute<gpu>", ShapeComputeGPU);

void SizeComputeGPU(const nnvm::NodeAttrs& attrs,
                    const OpContext& ctx,
                    const std::vector<TBlob>& inputs,
                    const std::vector<OpReqType>& req,
                    const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  using namespace mxnet_op;
  CHECK_EQ(inputs.size(), 1U);
  CHECK_EQ(outputs.size(), 1U);
  CHECK_EQ(req.size(), 1U);
  const TBlob& in_data = inputs[0];
  const TBlob& out_data = outputs[0];
  mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
  const index_t size_var = in_data.Size();
  hipMemcpyAsync(out_data.dptr_,
                  &size_var,
                  1U * sizeof(int64_t),
                  hipMemcpyHostToDevice,
                  mshadow::Stream<gpu>::GetStream(s));
}

NNVM_REGISTER_OP(size_array)
.set_attr<FCompute>("FCompute<gpu>", SizeComputeGPU);

NNVM_REGISTER_OP(Cast)
.set_attr<FCompute>("FCompute<gpu>", CastCompute<gpu>);

NNVM_REGISTER_OP(_backward_cast)
.set_attr<FCompute>("FCompute<gpu>", CastCompute<gpu>);

// negative
NNVM_REGISTER_OP(negative)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::negation>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::negation>);

// reciprocal
NNVM_REGISTER_OP(reciprocal)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::reciprocal>);

NNVM_REGISTER_OP(_backward_reciprocal)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::reciprocal_grad> >);

// abs
NNVM_REGISTER_OP(abs)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::abs>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::abs>);

NNVM_REGISTER_OP(_backward_abs)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<gpu, unary_bwd<mshadow_op::sign> >);

// sign
NNVM_REGISTER_OP(sign)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::sign>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::sign>);

NNVM_REGISTER_OP(_backward_sign)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::sign_grad> >);

// round
NNVM_REGISTER_OP(round)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::round>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::round>);

// ceil
NNVM_REGISTER_OP(ceil)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::ceil>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::ceil>);

// floor
NNVM_REGISTER_OP(floor)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::floor>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::floor>);

// trunc
NNVM_REGISTER_OP(trunc)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::trunc>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::trunc>);

// rint
NNVM_REGISTER_OP(rint)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::rint>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::rint>);


// fix
NNVM_REGISTER_OP(fix)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::fix>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::fix>);


// square
NNVM_REGISTER_OP(square)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::square>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::square>);

NNVM_REGISTER_OP(_backward_square)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::square_grad> >);

// sqrt
NNVM_REGISTER_OP(sqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::square_root>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::square_root>);


NNVM_REGISTER_OP(_backward_sqrt)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::square_root_grad> >);

// rsqrt
NNVM_REGISTER_OP(rsqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::reciprocal_square_root>);

NNVM_REGISTER_OP(_backward_rsqrt)
.set_attr<FCompute>("FCompute<gpu>",
  ElemwiseBinaryOp::Compute<gpu, unary_bwd<mshadow_op::reciprocal_square_root_grad> >);

// cbrt
NNVM_REGISTER_OP(cbrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::cube_root>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::cube_root>);


NNVM_REGISTER_OP(_backward_cbrt)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::cube_root_grad> >);

// rcbrt
NNVM_REGISTER_OP(rcbrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::reciprocal_cube_root>);

NNVM_REGISTER_OP(_backward_rcbrt)
.set_attr<FCompute>("FCompute<gpu>",
  ElemwiseBinaryOp::Compute<gpu, unary_bwd<mshadow_op::reciprocal_cube_root_grad> >);

// exp
NNVM_REGISTER_OP(exp)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::exp>);

// log
NNVM_REGISTER_OP(log)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::log>);

// log10
NNVM_REGISTER_OP(log10)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::log10>);

// log2
NNVM_REGISTER_OP(log2)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::log2>);

NNVM_REGISTER_OP(_backward_log)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::log_grad> >);

NNVM_REGISTER_OP(_backward_log10)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::log10_grad> >);

NNVM_REGISTER_OP(_backward_log2)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::log2_grad> >);

// log1p
NNVM_REGISTER_OP(log1p)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::log1p>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::log1p>);

NNVM_REGISTER_OP(_backward_log1p)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::log1p_grad> >);

// expm1
NNVM_REGISTER_OP(expm1)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::expm1>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::expm1>);

NNVM_REGISTER_OP(_backward_expm1)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::exp> >);

// gamma
NNVM_REGISTER_OP(gamma)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::gamma>);

NNVM_REGISTER_OP(_backward_gamma)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::gamma_grad> >);

// gammaln
NNVM_REGISTER_OP(gammaln)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::gammaln>);

NNVM_REGISTER_OP(_backward_gammaln)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::gammaln_grad> >);

// logical not
NNVM_REGISTER_OP(logical_not)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::nt>);

}  // namespace op
}  // namespace mxnet
