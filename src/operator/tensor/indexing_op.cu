#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file indexing_op.cu
 * \brief GPU implementation of indexing operator
 * \author Siyi Li, Chi Zhang
*/

#include "./indexing_op.h"
#include "./util/tensor_util-inl.cuh"
#include "./util/tensor_util-inl.h"

namespace mxnet {
namespace op {

/*! \brief If there are out-of-bound indices, out will be assigned to 1.
 */

struct is_valid_check {
  template<typename DType>
  MSHADOW_XINLINE static void Map(int i, char* out, const DType* data,
                                  const DType min, const DType max) {
    if (data[i] < min || data[i] > max) *out = 1;
  }
};


struct AddTakeGradRspGPUKernel {
  template<typename DType, typename IType>
  __device__ __forceinline__ static void Map(int tid,
                                             DType* out,
                                             const nnvm::dim_t* prefix_sum,
                                             const IType* data,
                                             const DType* ograd,
                                             const nnvm::dim_t row_length) {
    using nnvm::dim_t;
    const dim_t data_i = tid / row_length;
    const dim_t grad_i = tid % row_length;
    const dim_t irow = static_cast<dim_t>(data[data_i]);
    const dim_t rsp_row = prefix_sum[irow] - 1;
    const DType val = ograd[data_i * row_length + grad_i];
    atomicAdd(static_cast<DType *>(&(out[rsp_row*row_length+grad_i])), val);
  }
};

/*
 * \brief kernel for backward computation for take, executed with deterministic order
 * \param thread_id the thread id
 * \param out the output gradient data
 * \param lookup_table the table to lookup the position of an id in gradient array
 * \param sorted_data the sorted data input
 * \param original_idx the original indices of the sorted data input
 * \param ograd head gradient
 * \param row_length the output dimension
 * \param num_threads_per_row the number of threads to process a row together
 * \param SZ the number of features a thread is responsible for
 */
template<int SZ>
struct AddTakeGradRspDeterministicKernel {
  template<typename DType>
  __device__ __forceinline__ static void Map(int thread_id,
                                             DType* out,
                                             const nnvm::dim_t* lookup_table,
                                             const nnvm::dim_t* sorted_data,
                                             const nnvm::dim_t data_size,
                                             const nnvm::dim_t* original_idx,
                                             const DType* ograd,
                                             const nnvm::dim_t row_length,
                                             const nnvm::dim_t num_threads_per_row) {
    using nnvm::dim_t;
    int tid = thread_id / num_threads_per_row;
    const int feature_start = thread_id % num_threads_per_row * SZ;
    int num_features = SZ;
    if (feature_start + num_features > row_length) {
      num_features = row_length - feature_start;
    }
    if (tid == 0 || sorted_data[tid - 1] != sorted_data[tid]) {
      DType acc[SZ];
      #pragma unroll
      for (int i = 0; i < SZ; i++) {
        acc[i] = 0;
      }
      const dim_t data = sorted_data[tid];
      const dim_t row_id = lookup_table[data];
      const dim_t out_offset = row_id * row_length + feature_start;
      do {
        const dim_t idx = original_idx[tid];
        const dim_t ograd_offset = idx * row_length + feature_start;
        for (int i = 0; i < num_features; i++) {
          acc[i] += ograd[ograd_offset + i];
        }
        tid++;
      } while (tid < data_size && sorted_data[tid - 1] == sorted_data[tid]);
      for (int i = 0; i < num_features; i++) {
        out[out_offset + i] += acc[i];
      }
    }
  }
};

/*! \brief name the struct Take instead of take
 * to avoid conflict with the take function in mshadow
 */
template<bool clip = true>
struct TakeGPU {
  // assume that idx have been flattened to a 1-D tensor (N,)
  // assume that out_data and in_data have been flattened to 2-D tensors, (N, M) and (K, M)
  // M is the number of columns of in_data and out_data
  // K is the number of rows of in_data
  // i is the index of out_data
  template<typename DType, typename IType>
  MSHADOW_XINLINE static void Map(int i, DType* out_data, const DType* in_data,
                                  const IType* idx, const int64_t M, const int64_t K) {
    int64_t j = static_cast<int64_t>(idx[i/M]);
    if (clip) {
      if (j <= 0) j = 0;
      else if (j >= K) j = K - 1;
    } else {
      j = j % K;
      j += (j < 0) ? K : 0;
    }
    out_data[i] = in_data[j * M + i % M];
  }
};

/*
 * \brief returns true if all indices are between [min, max]
 * \param s the stream
 * \param data_ptr the indices on the stream
 * \param data_size the number of indices to examine
 * \param min the expected min value for indices
 * \param max the expected max value for indices
 * \param is_valid_ptr the temparary workspace
 */
template<typename DType>
bool CheckIndexOutOfBound(mshadow::Stream<gpu> *s, const DType* data_ptr, size_t data_size,
                          const DType min, const DType max, char* is_valid_ptr) {
  using namespace mxnet_op;
  int32_t is_valid = 0;
  Kernel<set_zero, gpu>::Launch(s, 1, is_valid_ptr);
  Kernel<is_valid_check, gpu>::Launch(s, data_size, is_valid_ptr, data_ptr, min, max);
  CUDA_CALL(hipMemcpy(&is_valid, is_valid_ptr, sizeof(char),
            hipMemcpyDeviceToHost));
  return is_valid == 0;
}

// Embedding forward implementation with dense weight
template<>
void EmbeddingOpForwardDnsImpl<gpu>(mshadow::Stream<gpu>* s,
                                    const TBlob& data,
                                    const TBlob& weight,
                                    const OpReqType req,
                                    const TBlob& output) {
  using namespace mxnet_op;
  const TShape& ishape = data.shape_;
  const TShape& oshape = output.shape_;

  MSHADOW_TYPE_SWITCH(output.type_flag_, DType, {
    MSHADOW_TYPE_SWITCH(data.type_flag_, IType, {
      Tensor<gpu, 1, IType> idx = data.get_with_shape<gpu, 1, IType>(
        Shape1(ishape.ProdShape(0, ishape.ndim())), s);
      Tensor<gpu, 2, DType> wmat = weight.get<gpu, 2, DType>(s);
      Tensor<gpu, 2, DType> out = output.get_with_shape<gpu, 2, DType>(
        Shape2(oshape.ProdShape(0, oshape.ndim()-1), oshape[oshape.ndim()-1]), s);
      Kernel<TakeGPU<true>, gpu>::Launch(s, oshape.Size(), out.dptr_, wmat.dptr_,
                                         idx.dptr_, wmat.shape_[1], wmat.shape_[0]);
    });
  });
}

template<>
void SparseEmbeddingOpForwardRspImpl<gpu>(const OpContext& ctx,
                                          const TBlob& data,
                                          const NDArray& weight,
                                          const OpReqType req,
                                          const TBlob& output) {
  if (req == kNullOp) return;
  using namespace rowsparse;
  using namespace mxnet_op;
  mshadow::Stream<gpu>* s = ctx.get_stream<gpu>();
  // zeros weight
  if (req == kWriteTo && !weight.storage_initialized()) {
    size_t out_size = output.shape_.Size();
    MSHADOW_TYPE_SWITCH(output.type_flag_, DType, {
      Fill<false>(s, TBlob(output.dptr<DType>(), mshadow::Shape1(out_size),
          gpu::kDevMask), kWriteTo, 0);
    })
    return;
  }
  // check out-of-bound indices
  MSHADOW_TYPE_SWITCH(data.type_flag_, DType, {
    DType min = 0;
    DType max = static_cast<DType>(weight.shape()[0] - 1);
    DType* data_ptr = data.dptr<DType>();
    size_t data_size = data.shape_.Size();
    Tensor<gpu, 1, char> workspace = ctx.requested[0]
        .get_space_typed<gpu, 1, char>(Shape1(1), s);
    char* is_valid_ptr = reinterpret_cast<char*>(workspace.dptr_);
    bool is_valid = CheckIndexOutOfBound(s, data_ptr, data_size, min, max, is_valid_ptr);
    CHECK(is_valid) << "SparseEmbedding input contains data out of bound";
  })
  // the weight is actually dense
  if (weight.aux_shape(kIdx)[0] == weight.shape()[0]) {
    EmbeddingOpForwardDnsImpl<gpu>(s, data, weight.data(), req, output);
  } else {
    EmbeddingOpForwardRspImpl<gpu>(s, data, weight, req, output);
  }
}

template<typename IType, typename DType, typename RType>
void SparseEmbeddingDeterministicKernelLaunch(const OpContext& ctx,
                                              const TBlob& ograd,
                                              const TBlob& data,
                                              const OpReqType req,
                                              const NDArray& output) {
  using namespace mshadow;
  using namespace mxnet_op;
  using namespace expr;
  using namespace rowsparse;
  using nnvm::dim_t;
  mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
  const dim_t num_rows = output.shape()[0];
  const dim_t row_length = output.shape()[1];
  const dim_t data_size = static_cast<dim_t>(data.shape_.Size());
  // temp resource declarations
  dim_t* lookup_table = NULL;
  void* temp_storage = NULL;
  dim_t* sorted_data = NULL;
  dim_t* original_idx = NULL;
  // calculate number of bytes for temp resources
  size_t lookup_table_bytes = num_rows * sizeof(dim_t);
  size_t sorted_data_storage_bytes = data_size * sizeof(dim_t);
  size_t original_idx_storage_bytes = data_size * sizeof(dim_t);
  size_t sort_workspace_size = SortByKeyWorkspaceSize<dim_t, dim_t, gpu>(data_size);
  size_t unique_workspace_bytes = 0;
  // estimate unique temp space
  IType* data_ptr = data.dptr<IType>();
  size_t *null_ptr = nullptr;
  // unique operations will be applied on sorted data
  hipcub::DeviceSelect::Unique(NULL, unique_workspace_bytes, sorted_data, sorted_data,
    null_ptr, data_size, Stream<gpu>::GetStream(s));
  // One more space reserved for unique count
  size_t temp_workspace_bytes = std::max(unique_workspace_bytes,
                                         sort_workspace_size);
  size_t total_storage_bytes = lookup_table_bytes + sorted_data_storage_bytes +
                               original_idx_storage_bytes + temp_workspace_bytes;

  // request resource and split it. layout is:
  // lookup_table, sorted_data, original_idx, temp_storage
  Tensor<gpu, 1, char> workspace = ctx.requested[0]
      .get_space_typed<gpu, 1, char>(Shape1(total_storage_bytes), s);
  lookup_table = reinterpret_cast<dim_t*>(workspace.dptr_);
  sorted_data = reinterpret_cast<dim_t*>(workspace.dptr_ + lookup_table_bytes);
  original_idx = reinterpret_cast<dim_t*>(workspace.dptr_ + lookup_table_bytes +
                                          sorted_data_storage_bytes);
  temp_storage = workspace.dptr_ + total_storage_bytes - temp_workspace_bytes;

  // check out-of-bound indices
  {
    IType min = 0;
    IType max = static_cast<IType>(output.shape()[0] - 1);
    IType* data_ptr = data.dptr<IType>();
    size_t data_size = data.shape_.Size();
    bool is_valid = CheckIndexOutOfBound(s, data_ptr, data_size, min, max,
                                         reinterpret_cast<char*>(temp_storage));
    CHECK(is_valid) << "Embedding input contains data out of bound";
  }

  // make a copy of the data, to be sorted
  TBlob sorted_data_blob(sorted_data, Shape1(data_size), gpu::kDevMask);
  auto sorted_data_tensor = sorted_data_blob.FlatTo1D<gpu, dim_t>(s);
  mxnet_op::copy(s, sorted_data_blob, data);

  // generate original idx
  Tensor<gpu, 1, dim_t> original_idx_tensor(original_idx, Shape1(data_size), s);
  Kernel<range_fwd, gpu>::Launch(s, data_size, 1, static_cast<dim_t>(0),
                                 static_cast<dim_t>(1), kWriteTo, original_idx);
  // sort data with its original idx
  int num_bits = common::ilog2ui(num_rows - 1);
  char* temp_storage_ptr = reinterpret_cast<char*>(temp_storage);
  Tensor<gpu, 1, char> temp_storage_tensor(temp_storage_ptr,
                                           Shape1(sort_workspace_size), s);
  SortByKey(sorted_data_tensor, original_idx_tensor, true,
            &temp_storage_tensor, 0, num_bits);

  // compute unique row ids based on sorted values.
  output.CheckAndAllocAuxData(kIdx, Shape1(data_size + 1));

  // fill row_idx array of output matrix, using the row_flg values
  RType* grad_row_idx = output.aux_data(kIdx).dptr<RType>();
  hipcub::DeviceSelect::Unique(temp_storage_ptr, unique_workspace_bytes, sorted_data,
      grad_row_idx, grad_row_idx + data_size, data_size, Stream<gpu>::GetStream(s));

  dim_t nnr = 0;
  CUDA_CALL(hipMemcpy(&nnr, grad_row_idx + data_size, sizeof(RType),
      hipMemcpyDeviceToHost));
  CHECK_EQ(output.shape().ndim(), 2) << "Unexcepted ndim";
  output.CheckAndAllocData(Shape2(nnr, output.shape()[1]));
  output.set_aux_shape(kIdx, Shape1(nnr));

  // generate lookup table
  Kernel<MarkLookupTable, gpu>::Launch(s, nnr, lookup_table, grad_row_idx);

  // accumulate gradients
  DType* grad_data = output.data().dptr<DType>();
  Fill<false>(s, TBlob(grad_data, Shape1(nnr * row_length), gpu::kDevMask),
              kWriteTo, 0);
  const int SZ = 4;
  const nnvm::dim_t num_threads_per_row = (row_length + SZ - 1) / SZ;
  Kernel<AddTakeGradRspDeterministicKernel<SZ>, gpu>::Launch(s, data_size * num_threads_per_row,
                     grad_data, lookup_table, sorted_data, data_size, original_idx,
                     ograd.dptr<DType>(), row_length, num_threads_per_row);
}

inline void SparseEmbeddingOpBackwardDeterministicRspImpl(const OpContext& ctx,
                                                          const TBlob& ograd,
                                                          const TBlob& data,
                                                          const OpReqType req,
                                                          const NDArray& output) {
  using nnvm::dim_t;
  if (req == kNullOp) return;
  CHECK_EQ(req, kWriteTo) << "SparseEmbedding layer doesn't support "
                          << "weight gradient calculation with req != write";

  mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
  const dim_t data_size = static_cast<dim_t>(data.shape_.Size());
  if (data_size == 0) {
    FillZerosRspImpl(s, output);
    return;
  }

  MSHADOW_TYPE_SWITCH(data.type_flag_, IType, {
    MSHADOW_TYPE_SWITCH(ograd.type_flag_, DType, {
      MSHADOW_IDX_TYPE_SWITCH(output.aux_type(rowsparse::kIdx), RType, {
        SparseEmbeddingDeterministicKernelLaunch<IType, DType, RType>(ctx, ograd, data,
                                                                      req, output);
      });
    });
  });
}


template<>
inline void SparseEmbeddingOpBackwardRspImpl<gpu>(const bool deterministic,
                                                  const OpContext& ctx,
                                                  const TBlob& ograd,
                                                  const TBlob& data,
                                                  const OpReqType req,
                                                  const NDArray& output) {
  if (deterministic) {
    SparseEmbeddingOpBackwardDeterministicRspImpl(ctx, ograd, data, req, output);
    return;
  }
  using namespace mshadow;
  using namespace mxnet_op;
  using namespace mshadow::expr;
  using namespace rowsparse;
  using nnvm::dim_t;
  if (req == kNullOp) return;
  CHECK_EQ(req, kWriteTo) << "SparseEmbedding layer doesn't support "
                          << "weight gradient calculation with req != write";

  // Request temporary storage for marking non-zero rows and prefix sum
  Stream<gpu> *s = ctx.get_stream<gpu>();
  dim_t num_rows = output.shape()[0];
  dim_t row_length = output.shape()[1];
  dim_t data_size = static_cast<dim_t>(data.shape_.Size());
  dim_t num_threads;

  MSHADOW_TYPE_SWITCH(data.type_flag_, IType, {
    MSHADOW_SGL_DBL_TYPE_SWITCH(ograd.type_flag_, DType, {
      MSHADOW_IDX_TYPE_SWITCH(output.aux_type(kIdx), RType, {
        dim_t* prefix_sum = NULL;
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      Stream<gpu>::GetStream(s));
        Tensor<gpu, 1, char> workspace = ctx.requested[0]
            .get_space_typed<gpu, 1, char>(Shape1(num_rows * sizeof(dim_t) +
                                           temp_storage_bytes), s);
        prefix_sum = reinterpret_cast<dim_t*>(workspace.dptr_);
        d_temp_storage = workspace.dptr_ + num_rows*sizeof(dim_t);
        num_threads = num_rows;
        Fill<false>(s, TBlob(prefix_sum, Shape1(num_threads), gpu::kDevMask), kWriteTo, 0);
        Kernel<MarkRowFlgKernel, gpu>::Launch(s, data_size, prefix_sum, data.dptr<IType>());

        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      mshadow::Stream<gpu>::GetStream(s));
        dim_t nnr = 0;
        CUDA_CALL(hipMemcpy(&nnr, &prefix_sum[num_rows-1], sizeof(dim_t),
            hipMemcpyDeviceToHost));
        if (nnr == 0) {
          FillZerosRspImpl(s, output);
          return;
        }
        output.CheckAndAlloc({Shape1(nnr)});
        RType* grad_row_idx = output.aux_data(kIdx).dptr<RType>();
        // fill row_idx array of output matrix, using the row_flg values
        Kernel<FillRspRowIdxKernel, gpu>::Launch(s, num_rows,
            grad_row_idx, prefix_sum, num_rows);
        // prefill with zeros
        DType* grad_data = output.data().dptr<DType>();
        Fill<false>(s, TBlob(grad_data, Shape1(nnr * row_length), gpu::kDevMask),
            kWriteTo, 0);
        // add the final gradients
        num_threads = row_length * data_size;
        Kernel<AddTakeGradRspGPUKernel, gpu>::Launch(s, num_threads, grad_data, prefix_sum,
            data.dptr<IType>(), ograd.dptr<DType>(), row_length);
      });
    });
  });
}

struct backward_gather_nd_gpu {
  template<typename DType, typename IType>
  MSHADOW_XINLINE static void Map(int i, int N, int M, int K,
                                  const mshadow::Shape<10> strides,
                                  DType* out, const DType* data,
                                  const IType* indices) {
    int offset = 0;
    for (int j = 0; j < M; ++j) {
      offset += strides[j] * static_cast<int>(indices[j*N + i]);
    }
    for (int j = 0; j < K; ++j) {
      atomicAdd(out + (offset + j), data[i * K + j]);
    }
  }
};

template<typename DType, typename IType>
inline void GatherNDBackwardImpl(int N, int M, int K,
                                 const mshadow::Shape<10> strides,
                                 DType* out,
                                 const DType* data,
                                 const IType* indices,
                                 mshadow::Stream<gpu> *s) {
  mxnet_op::Kernel<backward_gather_nd_gpu, gpu>::Launch(s, N, N, M, K, strides, out, data, indices);
}

template<>
void TakeOpForward<gpu>(const nnvm::NodeAttrs& attrs,
                        const OpContext& ctx,
                        const std::vector<TBlob>& inputs,
                        const std::vector<OpReqType>& req,
                        const std::vector<TBlob>& outputs) {
  using namespace mxnet_op;
  if (req[take_::kOut] == kNullOp) return;
  const TakeParam& param = nnvm::get<TakeParam>(attrs.parsed);
  CHECK_EQ(inputs.size(), 2U);
  CHECK_EQ(outputs.size(), 1U);

  const TShape& idxshape = inputs[take_::kIdx].shape_;
  const TShape& arrshape = inputs[take_::kArr].shape_;
  const TShape& oshape = outputs[take_::kOut].shape_;

  Stream<gpu> *s = ctx.get_stream<gpu>();
  const int actual_axis = param.axis + ((param.axis < 0) ? arrshape.ndim() : 0);

  MSHADOW_TYPE_SWITCH(outputs[0].type_flag_, DType, {  // output data type
    MSHADOW_TYPE_SWITCH(inputs[1].type_flag_, IType, {  // index data type
      if (actual_axis == 0) {
        if (param.mode == take_::kClip) {
          Kernel<TakeGPU<true>, gpu>::Launch(s, oshape.Size(),
                                             outputs[take_::kOut].dptr<DType>(),
                                             inputs[take_::kArr].dptr<DType>(),
                                             inputs[take_::kIdx].dptr<IType>(),
                                             oshape.Size()/idxshape.Size(), arrshape[0]);
        } else {
          Kernel<TakeGPU<false>, gpu>::Launch(s, oshape.Size(),
                                              outputs[take_::kOut].dptr<DType>(),
                                              inputs[take_::kArr].dptr<DType>(),
                                              inputs[take_::kIdx].dptr<IType>(),
                                              oshape.Size()/idxshape.Size(), arrshape[0]);
        }
      } else {
        mshadow::Shape<10> in_strides;
        int stride = 1;
        for (int i = arrshape.ndim() - 1; i >= 0; stride *= arrshape[i], --i) {
          in_strides[i] = stride;
        }
        mshadow::Shape<10> out_strides;
        stride = 1;
        for (int i = oshape.ndim() - 1; i >= 0; stride *= oshape[i], --i) {
          out_strides[i] = stride;
        }
        if (param.mode == take_::kClip) {
          Kernel<Take<true>, gpu>::Launch(s, oshape.Size(),
                                          outputs[take_::kOut].dptr<DType>(),
                                          inputs[take_::kArr].dptr<DType>(),
                                          inputs[take_::kIdx].dptr<IType>(),
                                          in_strides, out_strides, arrshape.ndim(), oshape.ndim(),
                                          idxshape.ndim(), arrshape[actual_axis], actual_axis);
        } else if (param.mode == take_::kWrap) {
          Kernel<Take<false>, gpu>::Launch(s, oshape.Size(),
                                           outputs[take_::kOut].dptr<DType>(),
                                           inputs[take_::kArr].dptr<DType>(),
                                           inputs[take_::kIdx].dptr<IType>(),
                                           in_strides, out_strides, arrshape.ndim(), oshape.ndim(),
                                           idxshape.ndim(), arrshape[actual_axis], actual_axis);
        }
      }
    });
  });
}

NNVM_REGISTER_OP(Embedding)
.set_attr<FCompute>("FCompute<gpu>", EmbeddingOpForward<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", SparseEmbeddingOpForwardEx<gpu>);

NNVM_REGISTER_OP(_contrib_SparseEmbedding)
.set_attr<FComputeEx>("FComputeEx<gpu>", SparseEmbeddingOpForwardEx<gpu>);

NNVM_REGISTER_OP(_backward_Embedding)
.set_attr<FCompute>("FCompute<gpu>", EmbeddingOpBackward<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", EmbeddingOpBackwardEx<gpu>);

NNVM_REGISTER_OP(_backward_SparseEmbedding)
.set_attr<FComputeEx>("FComputeEx<gpu>", SparseEmbeddingOpBackwardEx<gpu>);

NNVM_REGISTER_OP(take)
.set_attr<FCompute>("FCompute<gpu>", TakeOpForward<gpu>);

NNVM_REGISTER_OP(_backward_take)
.set_attr<FCompute>("FCompute<gpu>", TakeOpBackward<gpu>);

NNVM_REGISTER_OP(batch_take)
.set_attr<FCompute>("FCompute<gpu>", BatchTakeOpForward<gpu>);

NNVM_REGISTER_OP(one_hot)
.set_attr<FCompute>("FCompute<gpu>", OneHotOpForward<gpu>);

NNVM_REGISTER_OP(gather_nd)
.set_attr<FCompute>("FCompute<gpu>", GatherNDForward<gpu>);

NNVM_REGISTER_OP(scatter_nd)
.set_attr<FCompute>("FCompute<gpu>", ScatterNDForward<gpu>);

NNVM_REGISTER_OP(_backward_gather_nd)
.set_attr<FCompute>("FCompute<gpu>", GatherNDBackward<gpu>);

NNVM_REGISTER_OP(_scatter_set_nd)
.set_attr<FCompute>("FCompute<gpu>", ScatterSetNDForward<gpu>);
}  // namespace op
}  // namespace mxnet
